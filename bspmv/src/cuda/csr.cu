#include "csr.cuh"


__host__ void cuda_csrpack_matrixvector(block_csr &matrix, double* x, double* y) {
	unsigned int* d_irp;
	unsigned int* d_ja;
	double* d_as;
	double* d_x;
	double* d_y;

	hipMalloc((void**) &d_irp, matrix.getSizeIrp() * sizeof(unsigned int));
	hipMalloc((void**) &d_ja, matrix.getSizeJa() * sizeof(unsigned int));
	hipMalloc((void**) &d_as, matrix.getSizeAs() * sizeof(double));

	hipMalloc((void**) &d_y, (matrix.getRows() + matrix.getBlockHeight() - matrix.getRows() % matrix.getBlockHeight()) * sizeof(double));
	hipMalloc((void**) &d_x, matrix.getCols() * sizeof(double));

	hipMemcpy(d_irp, matrix.getCpuIrp(), matrix.getSizeIrp() * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_ja, matrix.getCpuJa(), matrix.getSizeJa() * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_as, matrix.getCpuAs(), matrix.getSizeAs() * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, matrix.getCols() * sizeof(double), hipMemcpyHostToDevice);
	//device_cuda_ellpack_matrixvector_simple_3x3<BLOCK_SIZE_X> <<<matrix.getBlockRows(), BLOCK_SIZE_X>>>(d_as, d_ja, d_x, d_y, matrix.getMaxBlocks());
	//printf("calling with<%d> <<<%d,%d,%xSize>>> (a,j,x,y,%d,%d,%d)\n",BLOCK_SIZE_X,matrix.getBlockRows(), BLOCK_SIZE_X, BLOCK_SIZE_X * matrix.getBlockHeight(),matrix.getMaxBlocks(), matrix.getBlockHeight(), matrix.getBlockWidth());


	hipMemcpy(y, d_y, matrix.getRows() * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_ja);
	hipFree(d_as);
	hipFree(d_y);
	hipFree(d_x);

	for (int var = 0; var < matrix.getRows(); ++var) {
		printf("%f,", y[var]);
	}
}

