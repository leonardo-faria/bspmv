#include "hip/hip_runtime.h"
#include "ell.cuh"

template<unsigned int blockSize, unsigned int blockEntryHeight>
__global__ void device_cuda_ellpack_matrixvector_simple(double* as, unsigned int* ja, double* x, unsigned int bew, unsigned int max_n_blocks) {

	__shared__ double sdata[blockSize * blockEntryHeight];
	unsigned int tid = threadIdx.x;
	unsigned int bid = threadIdx.x;

	int as_off = (bid * max_n_blocks +tid )* blockSize;
	int ja_end=(bid+1) * max_n_blocks;
	for (int ja_off = bid * max_n_blocks+tid; ja_off < max_n_blocks; ja_off += blockSize) {
		for (int i = 0; i < blockSize; ++i) {
			sdata[i / bew] += x[ja_off + i % bew] * as[as_off + i];
		}
		as_off+=blockSize;
	}
	/*
	 in
	 */
}

__host__ void cuda_ellpack_matrixvector(block_ell matrix, double* x, double* y) {
	unsigned int* d_ja;
	double* d_as;
	double* d_x;
	double* d_y;

	hipMalloc((void**) &d_ja, matrix.getSizeJa() * sizeof(unsigned int));
	hipMalloc((void**) &d_as, matrix.getSizeAs() * sizeof(double));

	hipMalloc((void**) &d_y, matrix.getRows() * sizeof(double));
	hipMalloc((void**) &d_x, matrix.getCols() * sizeof(double));

	hipMemcpy(d_ja, matrix.getCpuJa(), matrix.getSizeJa() * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_as, matrix.getCpuAs(), matrix.getSizeAs() * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, matrix.getCols() * sizeof(double), hipMemcpyHostToDevice);
	device_cuda_ellpack_matrixvector_simple<BLOCK_SIZE_X, BLOCK_ENTRY_H> <<<GRID_DIM, BLOCK_DIM>>>(d_as, d_ja, d_x, matrix.getBlockWidth(), matrix.getMaxBlocks());

	hipMemcpy(y, d_y, matrix.getRows() * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_ja);
	hipFree(d_as);
	hipFree(d_y);
	hipFree(d_x);

}
